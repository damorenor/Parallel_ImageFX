#include "hip/hip_runtime.h"
/************* add vector ******************************************************/
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

/*******************************************************************************/

__global__ void
vectorAdd(const int *A, char *B, int numElements, int width, int kernelSize)
{
    int k;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int aux = 0;
    int aux2 = 0;
   for(int p = 0; p < (width*3); p++)
   {
    if (i < numElements)
    {    
        if(k == 3){
            aux = A[(i*width*3)+ p]* -1 + A[(i*width*3)+p+3] * 0 + A[(i*width*3)+p+6] * 1 
                + A[(i+1)*(width*3) + p]* -2 + A[(i+1)*(width*3) + p+3] * 0 + A[(i+1)*(width*3) + p+6] * 2 
                + A[(i+2)*(width*3) + p]* -1 + A[(i+2)*(width*3) + p+3] * 0 + A[(i+2)*(width*3) + p+6] * 1;
            aux2= A[(i*width*3) + p]* -1 + A[(i*width*3) + p+3] * -2 + A[(i*width*3)+p+6] * -1 
                + A[(i+1)*(width*3) + p]* 0 + A[(i+1)*(width*3) + p+3] * 0 + A[(i+1)*(width*3) + p+6] * 0 
                + A[(i+2)*(width*3) + p]* 1 + A[(i+2)*(width*3) + p+3] * 2 + A[(i+2)*(width*3) + p+6] * 1;
            B[p+(i*width*3)] = (char) sqrt((float)(aux*aux + aux2*aux2));    
        }
        if(k == 5){
            aux = A[(i*width*3)+ p]* -2 + A[(i*width*3)+p+3] * -1 + A[(i*width*3)+p+6] * 0 + A[(i*width*3)+p+9] * 1 + A[(i*width*3)+p+12] * 2
                + A[(i+1)*(width*3) + p]* -2 + A[(i+1)*(width*3) + p+3] * -1 + A[(i+1)*(width*3) + p+6] * 0 + A[(i+1)*(width*3) + p+9] * 1 + A[(i+1)*(width*3) + p+12] * 2
                + A[(i+2)*(width*3) + p]* -4 + A[(i+2)*(width*3) + p+3] * -2 + A[(i+2)*(width*3) + p+6] * 0 + A[(i+2)*(width*3) + p+9] * 2 + A[(i+2)*(width*3) + p+12] * 4
                + A[(i+3)*(width*3) + p]* -2 + A[(i+3)*(width*3) + p+3] * -1 + A[(i+3)*(width*3) + p+6] * 0 + A[(i+3)*(width*3) + p+9] * 1 + A[(i+3)*(width*3) + p+12] * 2
                + A[(i+4)*(width*3) + p]* -2 + A[(i+4)*(width*3) + p+3] * -1 + A[(i+4)*(width*3) + p+6] * 0 + A[(i+4)*(width*3) + p+9] * 1 + A[(i+4)*(width*3) + p+12] * 2;
            aux2= A[(i*width*3)+ p]* -2 + A[(i*width*3)+p+3] * -2 + A[(i*width*3)+p+6] * -4 + A[(i*width*3)+p+9] * -2 + A[(i*width*3)+p+12] * -2
                + A[(i+1)*(width*3) + p]* -1 + A[(i+1)*(width*3) + p+3] * -1 + A[(i+1)*(width*3) + p+6] * -2 + A[(i+1)*(width*3) + p+9] * -1 + A[(i+1)*(width*3) + p+12] * -1  
                + A[(i+2)*(width*3) + p]* 0 + A[(i+2)*(width*3) + p+3] * 0 + A[(i+2)*(width*3) + p+6] * 0 + A[(i+2)*(width*3) + p+9] * 0 + A[(i+2)*(width*3) + p+12] * 0
                + A[(i+3)*(width*3) + p]* 1 + A[(i+3)*(width*3) + p+3] * 1 + A[(i+3)*(width*3) + p+6] * 2 + A[(i+3)*(width*3) + p+9] * 1 + A[(i+3)*(width*3) + p+12] * 1
                + A[(i+4)*(width*3) + p]* 2 + A[(i+4)*(width*3) + p+3] * 2 + A[(i+4)*(width*3) + p+6] * 4 + A[(i+4)*(width*3) + p+9] * 2 + A[(i+4)*(width*3) + p+12] * 2;

            B[p+(i*width*3)] = (char) sqrt((float)(aux*aux + aux2*aux2));   
        }
        //printf("hilo %d : %d %d %d %d %d %d %d %d %d con resultado x %d y %d = %d en la posición: %d\n",i, A[(i*width*3)+ p], A[(i*width*3)+p+3], A[(i*width*3)+p+6], A[(i+1)*width*3 + p], A[(i+1)*width*3 + p+3], A[(i+1)*width*3 + p+6], A[(i+2)*(width*3) + p], A[(i+2)*(width*3)+p+3], A[(i+2)*(width*3)+p+6],aux,aux2,B[p+(i*width*3)],p+(i*width*3));
        aux=0;
        aux2 = 0;
    }   
   }
}

/*******************************************************************************/
int main(void)
{
    //char *img2;
    char *name;
    char *nameOutput;
    int width, height, channels, kernel;
    struct timeval tval_before, tval_after, tval_result;

    name = (char *) malloc(sizeof(char)*20);
    nameOutput = (char *) malloc(sizeof(char)*20);

    printf("Introduce el nombre de la imagen, el nombre de la nueva imagen y el tamaño del kernel (3 o 5)  separados por un espacio: \n");
    scanf("%s %s %d", name, nameOutput, &kernel);
    if(kernel != 3 && kernel != 5){
        exit(1);
    }
    gettimeofday(&tval_before, NULL);    
    unsigned char *img = stbi_load(name, &width, &height, &channels, 3);
    if(img == NULL){
        printf("Error al Cargar Imagen: ");
        printf("%s\n", stbi_failure_reason());
        exit(1);
    }
    int pixel_row_num = width * channels;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = pixel_row_num*height;
    size_t size = numElements * sizeof(int);
    //img2 = (char *) malloc(numElements*sizeof(char));

    // Allocate the host input vector A
    int *h_A = (int *)malloc(size);

    // Allocate the host input vector B
    char *h_B = (char *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; i+=3)
    {
        //printf("%d ",img[i]);
        int t = img[i]*0.299 + img[i+1]*0.587 + img[i+2]*0.114;
        h_A[i] = t;
        h_A[i+1] = t;
        h_A[i+2] = t;
    }
    
    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    char *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // Copy the host input vectors A in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 128;
    int blocksPerGrid =(height + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, height, width, kernel);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*for (int i = 0; i < numElements; ++i)
    {
        //printf("%d ",img[i]);
        img2[i] = h_B[i];
    }*/
    /*for (int i = 0; i < numElements; ++i)
    {
        printf("%d ", h_B[i]);
    }*/
    
    // Free host memory
    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    stbi_write_jpg(nameOutput, width, height, channels, h_B, 100);
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    
    printf("t: %ld.%06lld\n", (long int)tval_result.tv_sec, (long long int)tval_result.tv_usec);
    
    free(nameOutput);
    free(name);
    free(h_A);
    free(h_B);
    return 0;
}
